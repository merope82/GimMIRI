#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "imMIRI.h"
#include "GimMIRI.cuh"
#include "externC.h"


void freeall(flux img_h,flux *img_d,input_h psf_h,input_d *psf_d,globalstruct *global){
    configdata *cfg = &cconfig;

    free(img_h.phi_x);
    free(img_h.phi_y);
    free(img_h.N);
    if (!psf_h.F)  free(psf_h.F);
    if (!psf_h.F0) free(psf_h.F0);
    if (!psf_h.x)  free(psf_h.x);
    if (!psf_h.y)  free(psf_h.y);

    for ( int i=0 ; i<cfg->ndev ; i++){
	gpuErrchk(hipSetDevice(cfg->devs[i]));
 	gpuErrchk(hipFree(img_d[i].phi_x));
	gpuErrchk(hipFree(img_d[i].phi_y));
	gpuErrchk(hipFree(img_d[i].N));
	gpuErrchk(hipFree(global[i].states));
	if (!psf_d[i].F) gpuErrchk(hipFree(psf_d[i].F));
        if (!psf_d[i].x) gpuErrchk(hipFree(psf_d[i].x));
        if (!psf_d[i].y) gpuErrchk(hipFree(psf_d[i].y));
    }
}